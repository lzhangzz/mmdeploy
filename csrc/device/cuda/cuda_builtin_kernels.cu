
#include <hip/hip_runtime.h>
// Copyright (c) OpenMMLab. All rights reserved.

namespace mmdeploy {
namespace cuda {

__global__ void FillKernel(void *dst, size_t dst_size, const void *pattern, size_t pattern_size) {
  size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

  auto p_dst = static_cast<uchar1 *>(dst);
  auto p_pattern = static_cast<const uchar1 *>(pattern);

  for (; idx < dst_size; idx += blockDim.x * gridDim.x) {
    auto ptr = idx % pattern_size;
    p_dst[idx] = p_pattern[ptr];
  }
}

int Fill(void *dst, size_t dst_size, const void *pattern, size_t pattern_size,
         hipStream_t stream) {
  const uint n_threads = 256;
  const uint n_blocks = (dst_size + n_threads - 1) / n_threads;

  FillKernel<<<n_blocks, n_threads, 0, stream>>>(dst, dst_size, pattern, pattern_size);

  return 0;
}

__global__ void AddKernel(const float *a, const float *b, float *c, int n) {
  for (auto i = threadIdx.x + blockIdx.x * blockDim.x; i < n; i += blockDim.x * gridDim.x) {
    c[i] = a[i] + b[i];
  }
}

__attribute__((visibility("default"))) //
void add(const float *a, const float *b, float *c, int n, void *stream) {
  constexpr int n_threads = 512;
  int n_blocks = (n + n_threads - 1) / n_threads;
  AddKernel<<<n_blocks, n_threads, 0, (hipStream_t)stream>>>(a, b, c, n);
}

}  // namespace cuda
}  // namespace mmdeploy
