#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include "codebase/mmocr/cuda/utils.h"
#include "thrust/iterator/counting_iterator.h"
#include "thrust/transform.h"

namespace mmdeploy::mmocr {

namespace panet {

struct _op {
  const float* logit;
  float* score;
  uint8_t* mask;
  float thr;
  __device__ void operator()(int index) const {
    float sigmoid = 1.f / (1.f + expf(-logit[index]));
    if (score) {
      score[index] = sigmoid;
    }
    mask[index] = sigmoid >= thr;
  }
};

void SigmoidAndThreshold(const float* d_logit, int n, float thr, uint8_t* d_mask, float* d_score,
                         hipStream_t stream) {
  thrust::counting_iterator<int> index{0};
  thrust::for_each_n(thrust::cuda::par.on(stream), index, n, _op{d_logit, d_score, d_mask, thr});
}

}  // namespace panet

namespace dbnet {

struct _op {
  float thr;
  __device__ bool operator()(float score) const { return score >= thr; }
};

void Threshold(const float* d_score, int n, float thr, uint8_t* d_mask, hipStream_t stream) {
  thrust::transform(thrust::cuda::par.on(stream), d_score, d_score + n, d_mask, _op{thr});
}

}  // namespace dbnet

}  // namespace mmdeploy::mmocr
